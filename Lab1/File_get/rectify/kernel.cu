#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include "lodepng.h"
#define MAX_THREAD 1024

__global__ void rectify(unsigned char * original_img, unsigned char* new_img, unsigned int num_thread, unsigned int id, unsigned int size) {
	int i = threadIdx.x + id;
	if(i < size)
		new_img[i] = (original_img[i] < 127 ? 127 : original_img[i]);
}

int main(int argc, char *argv[]) {
	if (argc != 4) {
		printf("Invalid number of arguments\n");
		return -1;
	}
	clock_t start = clock();

	unsigned char* original_img, *new_img;
	unsigned char* original_cudaImg, *new_cudaImg;

	unsigned int num_thread = atoi(argv[3]);
	if (num_thread > 1024) {
		printf("Maximum number of threads allowed is 1024. %d is too big\n", num_thread);
		return -1;
	}
	unsigned width, height;
	unsigned int imagesize;
	unsigned error;
	error = lodepng_decode32_file(&original_img, &width, &height,
		argv[1]);
	if (error) {
		printf("%d: %s\n", error, lodepng_error_text(error));
		return -1;
	}

	imagesize = width * height * 4 * sizeof(unsigned char);
	new_img = (unsigned char*)malloc(imagesize);

	hipMalloc((void**)&original_cudaImg, imagesize);
	hipMalloc((void**)&new_cudaImg, imagesize);
	hipMemcpy(original_cudaImg, original_img, imagesize, hipMemcpyHostToDevice);
	for (unsigned int i = 0; i < imagesize; i = i + num_thread) {
		rectify<<<1,num_thread>>>(original_cudaImg, new_cudaImg, num_thread, i, imagesize);
	}
	hipDeviceSynchronize();
	hipMemcpy(new_img, new_cudaImg, imagesize, hipMemcpyDeviceToHost);

	error = lodepng_encode32_file(argv[2], new_img, width, height);
	if (error) {
		printf("%d: %s\n", error, lodepng_error_text(error));
		return -1;
	}
	printf("%lu msec", clock() - start);

	hipFree(original_cudaImg);
	hipFree(new_cudaImg);
	
	return 0;
}